
# include <stdio.h>
# include<stdlib.h>
#include <hip/hip_runtime.h>

__global__ void convolution_1D_basic_kernel(int *inp, int *out,int *mask, int mask_width, int width){ 

    int i= blockDim.x* blockIdx.x+threadIdx.x;

    int inp_startPoint=i-(mask_width/2);

    int Conv_value=0;

    for(int j=0;j<mask_width;j++){
        if((inp_startPoint+j>=0) && (inp_startPoint+j)<=width){
            Conv_value+=inp[inp_startPoint+j]*mask[j];
        }
    }   

    out[i]=Conv_value;
}


int main(){
    int *inp, *out,*mask;
    int n;
    int mask_width;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    int size=n*sizeof(int);

    inp=(int *)malloc(size);
    out=(int * )malloc(size);
    
    printf("Enter Input Elements: ");
    for(int i=0;i<n;i++){
        scanf("%d",&inp[i]);
    }

    printf("Enter a odd size mask width: ");
    scanf("%d", &mask_width);



    mask=(int*) malloc(mask_width*sizeof(int));

    printf("Enter Mask Elements: ");
    for(int i=0;i<mask_width;i++){
        scanf("%d",&mask[i]);
    }

    int *din,*dout,*dmask;

    hipMalloc((void **)&din, size);
    hipMalloc((void **)&dout, size);

    hipMalloc((void**)&dmask,mask_width*sizeof(int) );

    

    hipMemcpy(din,inp, size, hipMemcpyHostToDevice);
    hipMemcpy(dout, out, size, hipMemcpyHostToDevice);
    hipMemcpy(dmask, mask, mask_width*sizeof(int) ,hipMemcpyHostToDevice);

    // call
    convolution_1D_basic_kernel<<<1, n>>>(din, dout, dmask,mask_width,n);


    hipMemcpy(out,dout,size,hipMemcpyDeviceToHost);
    printf("Final Array after convolution is :");
    for(int i=0;i<n;i++){
        printf("%d\t",out[i]);
    }

    hipFree(din);
    hipFree(dout);
    hipFree(dmask);
    free(inp);
    free(out);
    free(mask);

}