#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void octal(int *arr, int *ans, int n){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if(i < n){
        ans[i] = 0;
        while(arr[i] > 0){
            ans[i] = (arr[i] % 8) + (10 * ans[i]);
            arr[i] /= 8;
        }
        int val = 0;
        while(ans[i] > 0){
            val = (10 * val) + (ans[i] % 10);
            ans[i] /= 10;
        }
        ans[i] = val;
    }
}
__global__ void octal2(int *arr, int *ans, int n) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n) {
        int num = arr[i];
        char temp[16];  // store up to 16 octal digits
        int idx = 0;

        // Handle 0 explicitly
        if (num == 0) {
            temp[idx++] = '0';
        }

        // Convert to octal (in reverse)
        while (num > 0) {
            // 0 is converted to ascii, and is add to num%8 and then total ascii is converted to string
            temp[idx++] = (num % 8) + '0';  // char digit
            num /= 8;
        }

        // Convert reversed char array to correct integer
        int result = 0;
        for (int j = idx - 1; j >= 0; --j) {
            result = result * 10 + (temp[j] - '0');
        }

        ans[i] = result;
    }
}
// example of binary --> similar with octal
// 11          ans[i]=1101----->ans[i]=1011
// 11%2 11/2=5  -->1
// 5%2 5/2=2   -->1
// 2%2 2/2=1   -->0       
// 1%2 1/2=0   -->1

// 1011

int main(int argc, char **argv){
    int *arr, *ans, n;
    printf("enter size of array\n");
    scanf("%d", &n);
    arr = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for(int i = 0; i < n; ++i){
        scanf("%d", &arr[i]);
    }
    int *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));
    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    octal2<<<1, n>>>(darr, dans, n);
    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dans);
    hipFree(darr);

    printf("Octal Values of given array is :\t");
    for(int i = 0; i < n; ++i){
        printf("%d\t", ans[i]);
    }
    return 0;
}