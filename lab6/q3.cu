// %%writefile q3.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void evenPhase(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    if (tid % 2 == 0){
        i = tid;
    }
    else{
        i = tid + 1;
    }

    if (i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

__global__ void oddPhase(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    if (tid % 2 != 0){
        i = tid;
    }
    else{
        i = tid + 1;
    }
    if (i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

void oddEvenSort(int *arr, int n)
{
    int *d_arr;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    for (int i = 0; i < n; i++)
    {
        evenPhase<<<gridSize, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
        oddPhase<<<gridSize, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main()
{
    int *inp;
    int n;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    int size=n*sizeof(int);

    inp=(int *)malloc(size);
    
    printf("Enter Elements to be sorted: ");
    for(int i=0;i<n;i++){
        scanf("%d",&inp[i]);
    }

    oddEvenSort(inp, n);
    printf("\n");
    printf("Sorted Array is: ");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", inp[i]);
    }

    return 0;
}