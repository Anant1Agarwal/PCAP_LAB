#include "hip/hip_runtime.h"
# include <stdio.h>
# include<stdlib.h>
# include <hip/hip_runtime.h>

__global__ void ParallelSelectionSort(int *inp, int *out,int n){ 

    int tid= (blockDim.x*blockIdx.x) + threadIdx.x;

    if(tid<n){
        int data=inp[tid];
        int pos=0;

        for(int i=0;i<n;i++){
            //finding right position for element 
            //2nd conditon is for equaltiy, if that element is meant to be at tid position, it should come at tid position
            if((inp[i]<data) || (inp[i]==data && i<tid)){
                pos+=1;
            }
        }
        out[pos]=data;
    }

}

int main(){
    int *inp, *out;
    int n;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    int size=n*sizeof(int);

    inp=(int *)malloc(size);
    out=(int * )malloc(size);
    
    printf("Enter Elements to be sorted: ");
    for(int i=0;i<n;i++){
        scanf("%d",&inp[i]);
    }

    int *din,*dout;

    hipMalloc((void **)&din, size);
    hipMalloc((void **)&dout, size)

    hipMemcpy(din,inp, size, hipMemcpyHostToDevice);
    hipMemcpy(dout, out, size, hipMemcpyHostToDevice);


    dim3 dimGrid(ceil(n/256.0),1,1);
    dim3 dimBlock(256,1,1);
    // call
    ParallelSelectionSort<<<dimGrid,dimBlock>>>(din,dout, n);

    hipMemcpy(out,dout,size,hipMemcpyDeviceToHost);
    printf("Final Array after Sorting is :");
    for(int i=0;i<n;i++){
        printf("%d\t",out[i]);
    }

    hipFree(din);
    hipFree(dout);

    free(inp);
    free(out);

}
