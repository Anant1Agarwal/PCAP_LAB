#include "hip/hip_runtime.h"
// %%writefile add_q1.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void calculateY(double* x, double*y,int m, int c, int n){

    int i=blockDim.x*blockIdx.x+threadIdx.x;

    if(i<n){
       y[i]=m*x[i]+c;
    }
}

int main(){
    int n;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    double *x,*y;
    int m, c;


    x=(double *)malloc(n*sizeof(double));
    y=(double *)malloc(n*sizeof(double));

    printf("\n Enter Array X: ");
    
    for(int i=0;i<n;i++){
        scanf("%lf", &x[i]);
    }
    printf("Enter Slope and intercept: ");
    scanf("%d %d", &m, &c)

    double *dy,*dx;
   
    int size=n*sizeof(double);

    hipMalloc((void **)&dy,size);
    hipMalloc((void**)&dx,size);
  
    hipMemcpy(dy, y, size,hipMemcpyHostToDevice);
    hipMemcpy(dx, x, size,hipMemcpyHostToDevice);

    calculateY<<<1,n>>>(dx,dy,m,c,n);

    hipMemcpy(y,dy,size,hipMemcpyDeviceToHost);

    printf("\nFinal Array Y=mx+c i.e Y=(%d)*(x)+%d is : ",m,c);
    
    for(int i=0;i<n;i++){
        printf("%d\t",y[i]);
    }

    free(x);
    free(y);
 
    hipFree(dx);
    hipFree(dy)

}