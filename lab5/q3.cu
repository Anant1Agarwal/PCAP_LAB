// %%writefile q3.cu
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void ConvertToSines(double* a,int n){
    int i=blockDim.x*blockIdx.x+threadIdx.x;

    if(i<n){
       a[i]=sin(a[i]);
    }
}

int main(){
    double*a, *da;
    int n; 
    printf("Enter Number of Elements :");
    scanf("%d",&n);

    int size=n*sizeof(double);
    a=(double*)malloc(size);

    printf("Enter Elements: ");
    for(int i=0;i<n;i++){
        scanf("%lf",&a[i]);
    }

    hipMalloc((void**)&da,size);
    hipMemcpy(da,a, size, hipMemcpyHostToDevice);

    ConvertToSines<<<1,n>>>(da,n);

    hipMemcpy(a,da,size,hipMemcpyDeviceToHost);

    printf("\nSines of Elements are: ");
    for(int i=0;i<n;i++){
        printf("%lf\t",a[i]);
    }

    hipFree(da);
    free(a);
}