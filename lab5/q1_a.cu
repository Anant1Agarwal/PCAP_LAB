// %%writefile q1_a.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void vecAddKernel(int *A, int *B, int *C,int n){
    int i=blockDim.x*blockIdx.x+threadIdx.x;

    printf("In Kernel function, i=%d ",i);
    if(i<n){
        C[i]=A[i]+B[i];
    }
}

int main(){
    int n;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    int *a,*b,*c;

    a=(int *)malloc(n*sizeof(int));
    b=(int *)malloc(n*sizeof(int));
    c=(int *)malloc(n*sizeof(int));

    printf("\n Enter Array A: ");
    
    for(int i=0;i<n;i++){
        scanf("%d", &a[i]);
    }

    printf("\n Enter Array B: ");
    
    for(int i=0;i<n;i++){
        scanf("%d", &b[i]);
    }

    int *da, *db, *dc;

    int size=n*sizeof(int);

    //part-1
    hipMalloc((void **)&da,size);
    hipMalloc((void **)&db,size);
    hipMalloc((void **)&dc,size);

    hipMemcpy(da, a, size,hipMemcpyHostToDevice);
    hipMemcpy(db, b, size,hipMemcpyHostToDevice);

    // part-2 blocks and threads
    // int blocks = (n + 255) / 256;
    // vecAddKernel<<<blocks, 256>>>(da, db, dc, n);
    // OR
    // vecAddKernel<<< ceil(n/256.0), 256 >>>(da, db, dc, n);
    // OR 
    // 1 block and n threads
    vecAddKernel<<<1,n>>>(da,db,dc,n);
    
    // Error check
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // part-3 transfer to C

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    printf("\nFinal Array after adding both is : ");
    
    for(int i=0;i<n;i++){
        printf("%d\t",c[i]);
    }

    free(a);
    free(b);
    free(c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

// / !nvcc q1.cu -o output.out

// / !./output.out


// Enter Number of Elements: 4

//  Enter Array A: 3 54 3 3

//  Enter Array B: 2 5 6 2
// In Kernel function, i=192 In Kernel function, i=193 In Kernel function, i=194 In Kernel function, i=195 In Kernel function, i=196 In Kernel function, i=197 In Kernel function, i=198 In Kernel function, i=199 In Kernel function, i=200 In Kernel function, i=201 In Kernel function, i=202 In Kernel function, i=203 In Kernel function, i=204 In Kernel function, i=205 In Kernel function, i=206 In Kernel function, i=207 In Kernel function, i=208 In Kernel function, i=209 In Kernel function, i=210 In Kernel function, i=211 In Kernel function, i=212 In Kernel function, i=213 In Kernel function, i=214 In Kernel function, i=215 In Kernel function, i=216 In Kernel function, i=217 In Kernel function, i=218 In Kernel function, i=219 In Kernel function, i=220 In Kernel function, i=221 In Kernel function, i=222 In Kernel function, i=223 In Kernel function, i=96 In Kernel function, i=97 In Kernel function, i=98 In Kernel function, i=99 In Kernel function, i=100 In Kernel function, i=101 In Kernel function, i=102 In Kernel function, i=103 In Kernel function, i=104 In Kernel function, i=105 In Kernel function, i=106 In Kernel function, i=107 In Kernel function, i=108 In Kernel function, i=109 In Kernel function, i=110 In Kernel function, i=111 In Kernel function, i=112 In Kernel function, i=113 In Kernel function, i=114 In Kernel function, i=115 In Kernel function, i=116 In Kernel function, i=117 In Kernel function, i=118 In Kernel function, i=119 In Kernel function, i=120 In Kernel function, i=121 In Kernel function, i=122 In Kernel function, i=123 In Kernel function, i=124 In Kernel function, i=125 In Kernel function, i=126 In Kernel function, i=127 In Kernel function, i=0 In Kernel function, i=1 In Kernel function, i=2 In Kernel function, i=3 In Kernel function, i=4 In Kernel function, i=5 In Kernel function, i=6 In Kernel function, i=7 In Kernel function, i=8 In Kernel function, i=9 In Kernel function, i=10 In Kernel function, i=11 In Kernel function, i=12 In Kernel function, i=13 In Kernel function, i=14 In Kernel function, i=15 In Kernel function, i=16 In Kernel function, i=17 In Kernel function, i=18 In Kernel function, i=19 In Kernel function, i=20 In Kernel function, i=21 In Kernel function, i=22 In Kernel function, i=23 In Kernel function, i=24 In Kernel function, i=25 In Kernel function, i=26 In Kernel function, i=27 In Kernel function, i=28 In Kernel function, i=29 In Kernel function, i=30 In Kernel function, i=31 In Kernel function, i=224 In Kernel function, i=225 In Kernel function, i=226 In Kernel function, i=227 In Kernel function, i=228 In Kernel function, i=229 In Kernel function, i=230 In Kernel function, i=231 In Kernel function, i=232 In Kernel function, i=233 In Kernel function, i=234 In Kernel function, i=235 In Kernel function, i=236 In Kernel function, i=237 In Kernel function, i=238 In Kernel function, i=239 In Kernel function, i=240 In Kernel function, i=241 In Kernel function, i=242 In Kernel function, i=243 In Kernel function, i=244 In Kernel function, i=245 In Kernel function, i=246 In Kernel function, i=247 In Kernel function, i=248 In Kernel function, i=249 In Kernel function, i=250 In Kernel function, i=251 In Kernel function, i=252 In Kernel function, i=253 In Kernel function, i=254 In Kernel function, i=255 In Kernel function, i=128 In Kernel function, i=129 In Kernel function, i=130 In Kernel function, i=131 In Kernel function, i=132 In Kernel function, i=133 In Kernel function, i=134 In Kernel function, i=135 In Kernel function, i=136 In Kernel function, i=137 In Kernel function, i=138 In Kernel function, i=139 In Kernel function, i=140 In Kernel function, i=141 In Kernel function, i=142 In Kernel function, i=143 In Kernel function, i=144 In Kernel function, i=145 In Kernel function, i=146 In Kernel function, i=147 In Kernel function, i=148 In Kernel function, i=149 In Kernel function, i=150 In Kernel function, i=151 In Kernel function, i=152 In Kernel function, i=153 In Kernel function, i=154 In Kernel function, i=155 In Kernel function, i=156 In Kernel function, i=157 In Kernel function, i=158 In Kernel function, i=159 In Kernel function, i=160 In Kernel function, i=161 In Kernel function, i=162 In Kernel function, i=163 In Kernel function, i=164 In Kernel function, i=165 In Kernel function, i=166 In Kernel function, i=167 In Kernel function, i=168 In Kernel function, i=169 In Kernel function, i=170 In Kernel function, i=171 In Kernel function, i=172 In Kernel function, i=173 In Kernel function, i=174 In Kernel function, i=175 In Kernel function, i=176 In Kernel function, i=177 In Kernel function, i=178 In Kernel function, i=179 In Kernel function, i=180 In Kernel function, i=181 In Kernel function, i=182 In Kernel function, i=183 In Kernel function, i=184 In Kernel function, i=185 In Kernel function, i=186 In Kernel function, i=187 In Kernel function, i=188 In Kernel function, i=189 In Kernel function, i=190 In Kernel function, i=191 In Kernel function, i=32 In Kernel function, i=33 In Kernel function, i=34 In Kernel function, i=35 In Kernel function, i=36 In Kernel function, i=37 In Kernel function, i=38 In Kernel function, i=39 In Kernel function, i=40 In Kernel function, i=41 In Kernel function, i=42 In Kernel function, i=43 In Kernel function, i=44 In Kernel function, i=45 In Kernel function, i=46 In Kernel function, i=47 In Kernel function, i=48 In Kernel function, i=49 In Kernel function, i=50 In Kernel function, i=51 In Kernel function, i=52 In Kernel function, i=53 In Kernel function, i=54 In Kernel function, i=55 In Kernel function, i=56 In Kernel function, i=57 In Kernel function, i=58 In Kernel function, i=59 In Kernel function, i=60 In Kernel function, i=61 In Kernel function, i=62 In Kernel function, i=63 In Kernel function, i=64 In Kernel function, i=65 In Kernel function, i=66 In Kernel function, i=67 In Kernel function, i=68 In Kernel function, i=69 In Kernel function, i=70 In Kernel function, i=71 In Kernel function, i=72 In Kernel function, i=73 In Kernel function, i=74 In Kernel function, i=75 In Kernel function, i=76 In Kernel function, i=77 In Kernel function, i=78 In Kernel function, i=79 In Kernel function, i=80 In Kernel function, i=81 In Kernel function, i=82 In Kernel function, i=83 In Kernel function, i=84 In Kernel function, i=85 In Kernel function, i=86 In Kernel function, i=87 In Kernel function, i=88 In Kernel function, i=89 In Kernel function, i=90 In Kernel function, i=91 In Kernel function, i=92 In Kernel function, i=93 In Kernel function, i=94 In Kernel function, i=95 
// Final Array after adding both is : 5	59	9	5	
