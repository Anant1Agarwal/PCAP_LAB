// %%writefile q1_b.cu

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vecAddKernel(int *A, int *B, int *C,int n){
    int i=blockDim.x*blockIdx.x+threadIdx.x;

    printf("In Kernel function, i=%d ",i);
    if(i<n){
        C[i]=A[i]+B[i];
    }
}

int main(){
    int n;
    printf("Enter Number of Elements: ");
    scanf("%d",&n);

    int *a,*b,*c;

    a=(int *)malloc(n*sizeof(int));
    b=(int *)malloc(n*sizeof(int));
    c=(int *)malloc(n*sizeof(int));

    printf("\n Enter Array A: ");
    
    for(int i=0;i<n;i++){
        scanf("%d", &a[i]);
    }

    printf("\n Enter Array B: ");
    
    for(int i=0;i<n;i++){
        scanf("%d", &b[i]);
    }

    int *da, *db, *dc;

    int size=n*sizeof(int);

    //part-1
    hipMalloc((void **)&da,size);
    hipMalloc((void **)&db,size);
    hipMalloc((void **)&dc,size);

    hipMemcpy(da, a, size,hipMemcpyHostToDevice);
    hipMemcpy(db, b, size,hipMemcpyHostToDevice);

    // 1 block and n threads
    // vecAddKernel<<<1,n>>>(da,db,dc,n);

    //n blocks and 1 thread
    vecAddKernel<<<n,1>>>(da,db,dc,n);

    
    // Error check
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // part-3 transfer to C

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    printf("\nFinal Array after adding both is : ");
    
    for(int i=0;i<n;i++){
        printf("%d\t",c[i]);
    }

    free(a);
    free(b);
    free(c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

// / !nvcc q1.cu -o output.out

// / !./output.out
